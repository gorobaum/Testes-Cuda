#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

__global__ void MatrixCopy (double* MatrixA, double* MatrixB, int row, int column) {
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  MatrixB[i*column+j] = MatrixA[i*column+j];
}

int main () {
  int row = 32,
      column = 32,
      i = 0,
      j = 0;
  dim3 threadPerBlock(16, 16),
       blocksPerGrid(row/threadPerBlock.x, column/threadPerBlock.y);
  size_t size = row*column*sizeof(double);
  double *MatrixA, *MatrixB, *cudaMA, *cudaMB;
  
  /* Matrix allocation. */
  MatrixA = (double*)malloc(size);
  MatrixB = (double*)malloc(size);
  
  for (i = 0; i < row; i++)
    for (j = 0; j < column; j++) {
      MatrixA[i*column+j] = i*column+j;
      MatrixB[i*column+j] = 0.0;
    }
 
  /* Cuda memory allocation. */
  if (hipMalloc(&cudaMA, size) != hipSuccess)
      printf("Erro na alçocação de recursos!\n");
  if (hipMalloc(&cudaMB, size) != hipSuccess)
      printf("Erro na alçocação de recursos!\n");

  /* Cuda memory copy. */
  if (hipMemcpy(cudaMA, MatrixA, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");
  if (hipMemcpy(cudaMB, MatrixB, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");

  /* Cuda kernel call. */
  MatrixCopy<<<blocksPerGrid, threadPerBlock>>>(cudaMA, cudaMB, row, column);
  hipDeviceSynchronize();

  if (hipMemcpy(MatrixB, cudaMB, size, hipMemcpyDeviceToHost) != hipSuccess)
      printf("Erro na cópia do Device para o Host!\n");

  for (i = 0; i < row; i++) {
    for (j = 0; j < column; j++) {
      printf("%5.lf ", MatrixB[i*column+j]);
    }
    printf("\n");
  }
  
  hipFree(&cudaMA);
  hipFree(&cudaMB);

  return 0;
}
