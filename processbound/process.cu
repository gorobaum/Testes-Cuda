#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

__global__ void MatrixCopy (float* MatrixA, float* MatrixB, float* MatrixC, int N) {
  int j = blockIdx.x*blockDim.x+threadIdx.x;
  int i = blockIdx.y*blockDim.y+threadIdx.y;
  int k;
  MatrixC[i*N+j] = 0;
  for (k = 0; k < N; k++ )
    MatrixC[i*N+j] += MatrixA[i*N+k]*MatrixB[k*N+j];
}

int main () {
  int N = 500,
      i = 0,
      j = 0;
  dim3 threadPerBlock(32, 32),
       blocksPerGrid(N/threadPerBlock.x+1, N/threadPerBlock.y+1);
  size_t size = N*N*sizeof(float);
  float *MatrixA, *MatrixB, *MatrixC, *cudaMA, *cudaMB, *cudaMC;
  float time;
  hipEvent_t start, stop;
  
  /* Matrix allocation. */
  MatrixA = (float*)malloc(size);
  MatrixB = (float*)malloc(size);
  MatrixC = (float*)malloc(size);
 
  for (i = 0; i < N; i++)
    for (j = 0; j < N; j++) {
      MatrixA[i*N+j] = i+j;
      MatrixB[i*N+j] = 2.0;
      MatrixC[i*N+j] = 0.0;
    }
 
  /* Cuda memory allocation. */
  if (hipMalloc(&cudaMA, size) != hipSuccess)
      printf("Erro na alocação de recursos!\n");
  if (hipMalloc(&cudaMB, size) != hipSuccess)
      printf("Erro na alocação de recursos!\n");
  if (hipMalloc(&cudaMC, size) != hipSuccess)
      printf("Erro na alocação de recursos!\n");

  /* Cuda memory copy. */
  if (hipMemcpy(cudaMA, MatrixA, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");
  if (hipMemcpy(cudaMB, MatrixB, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");
  if (hipMemcpy(cudaMC, MatrixC, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");

  /* Cuda time counter init. */
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Cuda kernel call. */
  hipEventRecord(start, 0);
  MatrixCopy<<<blocksPerGrid, threadPerBlock>>>(cudaMA, cudaMB, cudaMC, N);
  hipDeviceSynchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize( stop );

  /* Calculating run time. */
  hipEventElapsedTime( &time, start, stop );
  hipEventDestroy( start );
  hipEventDestroy( stop );
  printf("Total run time on GPU = %fms\n", time);

  if (hipMemcpy(MatrixC, cudaMC, size, hipMemcpyDeviceToHost) != hipSuccess)
      printf("Erro na cópia do Device para o Host!\n");

  
  for (i = 0; i < N; i++) {
    for (j = 0; j < N; j++) {
      printf("C[%d][%d] = %f\n", i, j, MatrixC[i*N+j]);
    }
  }
  
  hipFree(&cudaMA);
  hipFree(&cudaMB);

  return 0;
}
