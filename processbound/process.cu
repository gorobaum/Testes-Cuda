#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

__global__ void MatrixCopy (double* MatrixA, double* MatrixB, double* MatrixC, int N) {
  int j = blockIdx.x*blockDim.x+threadIdx.x;
  int i = blockIdx.y*blockDim.y+threadIdx.y;
  int k;

  for (k = 0; k < N; k++ )
    MatrixC[i*N+j] += MatrixA[i*N+k]*MatrixB[k+j*N];
}

int main () {
  int N = 32,
      i = 0,
      j = 0;
  dim3 threadPerBlock(16, 16),
       blocksPerGrid(N/threadPerBlock.x, N/threadPerBlock.y);
  size_t size = N*N*sizeof(double);
  double *MatrixA, *MatrixB, *MatrixC, *cudaMA, *cudaMB, *cudaMC;
  
  /* Matrix allocation. */
  MatrixA = (double*)malloc(size);
  MatrixB = (double*)malloc(size);
  MatrixC = (double*)malloc(size);
 
  for (i = 0; i < N; i++)
    for (j = 0; j < N; j++) {
      MatrixA[i*N+j] = i*N+j;
      MatrixB[i*N+j] = 2.0;
      MatrixC[i*N+j] = 0.0;
    }
 
  /* Cuda memory allocation. */
  if (hipMalloc(&cudaMA, size) != hipSuccess)
      printf("Erro na alçocação de recursos!\n");
  if (hipMalloc(&cudaMB, size) != hipSuccess)
      printf("Erro na alçocação de recursos!\n");
  if (hipMalloc(&cudaMC, size) != hipSuccess)
      printf("Erro na alçocação de recursos!\n");

  /* Cuda memory copy. */
  if (hipMemcpy(cudaMA, MatrixA, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");
  if (hipMemcpy(cudaMB, MatrixB, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");
  if (hipMemcpy(cudaMC, MatrixC, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");

  /* Cuda kernel call. */
  MatrixCopy<<<blocksPerGrid, threadPerBlock>>>(cudaMA, cudaMB, cudaMC, N);
  hipDeviceSynchronize();

  if (hipMemcpy(MatrixC, cudaMC, size, hipMemcpyDeviceToHost) != hipSuccess)
      printf("Erro na cópia do Device para o Host!\n");

  for (i = 0; i < N; i++) {
    for (j = 0; j < N; j++) {
      printf("%5.lf\n", MatrixC[i*N+j]);
    }
    printf("\n");
  }
  
  hipFree(&cudaMA);
  hipFree(&cudaMB);

  return 0;
}
