#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "memory_kernel.h"

__global__ void copy_kernel(double *ma, double *mb) {
  int row = threadIdx.x;
  int column = threadIdx.y;

  row *= MS;
  mb[row+column] = 0.0;
 
  mb[row+column] = ma[row+column];
  if (row == 0 && column == 0) printf("MA[0] = %f\n", mb[row+column]);
}

void copyMatrix_caller(double *Ma, double *Mb) {
  double *cudamtxa, *cudamtxb;
  dim3 threadPerBlock(MS,MS);

  hipMalloc(&cudamtxa, MS*MS*sizeof(double));
  hipMalloc(&cudamtxb, MS*MS*sizeof(double));

  hipMemcpy(cudamtxa, Ma, MS*MS*sizeof(double), hipMemcpyHostToDevice);

  copy_kernel<<<1, threadPerBlock>>>(cudamtxa, cudamtxb);
  hipDeviceSynchronize();
  hipMemcpy(Mb, cudamtxb, MS*MS*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
}
