#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "memory_kernel.h"

__global__ void multi_kernel(double *ma, double *mb, double *mc) {
  int row = threadIdx.x;
  int column = threadIdx.y;
  int i = 0;
  
  row *= MS;
  mc[row+column] = 0;
  
  for (i = 0; i < MS; i++) {
    mc[row+column] += ma[row+i]*mb[column+i*MS];
  }
}

void matrixMulti_caller(double *Ma, double *Mb, double *Mc) {
  double *cudamtxa, *cudamtxb, *cudamtxc;
  dim3 threadPerBlock(MS,MS);
  size_t free, total;
  
  hipDeviceReset();

  hipMemGetInfo(&free, &total);
  printf("GPU Memory Info -\n");
  printf("GPU Free Memory = %d MB\n", free/(1024*1024));
  printf("GPU Total Memory = %d MB\n", total/(1024*1024));
  getchar();

  hipMalloc(&cudamtxa, MS*MS*sizeof(double));
  hipMalloc(&cudamtxb, MS*MS*sizeof(double));
  hipMalloc(&cudamtxc, MS*MS*sizeof(double));

  hipMemcpy(cudamtxa, Ma, MS*MS*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(cudamtxb, Mb, MS*MS*sizeof(double), hipMemcpyHostToDevice);

  multi_kernel<<<1, threadPerBlock>>>(cudamtxa, cudamtxb, cudamtxc);


  hipMemcpy(Mc, cudamtxc, MS*MS*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
}
