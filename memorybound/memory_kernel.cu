#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "memory_kernel.h"

__global__ void copy_kernel(double *ma, double *mb) {
  int row = threadIdx.x;
  int column = threadIdx.y;
  
  row *= MS;
  mb[row+column] = 0.0;
}  

void copyMatrix_caller(double *Ma, double *Mb) {
  double *cudamtxa, *cudamtxb;
  dim3 threadPerBlock(MS,MS);
  int i;

  for (i = 0; i < MS*MS; i++) if (Mb[i] != 0.0) printf("Erro!%d\n", i);
  if (hipMalloc(&cudamtxa, MS*MS*sizeof(double)) == hipErrorOutOfMemory) {
    printf("Erro na alocação de memória do CUDA.\n");
    exit(0);
  }
  if (hipMalloc(&cudamtxb, MS*MS*sizeof(double)) == hipErrorOutOfMemory) {
    printf("Erro na alocação de memória do CUDA.\n");
    exit(0);
  }

  if (hipMemcpy(cudamtxa, Ma, MS*MS*sizeof(double), hipMemcpyHostToDevice) != hipSuccess) {
    printf("erro na copia de memória.\n");
    exit(0);
  }

  copy_kernel<<<1, threadPerBlock>>>(cudamtxa, cudamtxb);
  if (hipMemcpy(Ma, cudamtxa, MS*MS*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess) {
    printf("erro na copia de memória.\n");
    exit(0);
  }
  for (i = 0; i < MS*MS; i++) printf("Ma[%d] = %f\n", i, Ma[i]);
  hipDeviceSynchronize();
}
