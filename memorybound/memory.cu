#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

__global__ void MatrixCopy (float* MatrixA, float* MatrixB, int row, int column) {
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  MatrixB[i*column+j] = MatrixA[i*column+j];
}

int main () {
  int row = 500,
      column = 500,
      i = 0,
      j = 0;
  dim3 threadPerBlock(16, 16),
       blocksPerGrid(row/threadPerBlock.x+1, column/threadPerBlock.y+1);
  size_t size = row*column*sizeof(float);
  float *MatrixA, *MatrixB, *cudaMA, *cudaMB;
  float time;
  hipEvent_t start, stop;
  
  /* Matrix allocation. */
  MatrixA = (float*)malloc(size);
  MatrixB = (float*)malloc(size);
  
  for (i = 0; i < row; i++)
    for (j = 0; j < column; j++) {
      MatrixA[i*column+j] = i+j;
      MatrixB[i*column+j] = 0.0;
    }
 
  /* Cuda memory allocation. */
  if (hipMalloc(&cudaMA, size) != hipSuccess)
      printf("Erro na alçocação de recursos!\n");
  if (hipMalloc(&cudaMB, size) != hipSuccess)
      printf("Erro na alçocação de recursos!\n");

  /* Cuda memory copy. */
  if (hipMemcpy(cudaMA, MatrixA, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");
  if (hipMemcpy(cudaMB, MatrixB, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");
  
  /* Cuda time counter init. */
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Cuda kernel call. */
  hipEventRecord(start, 0);
  MatrixCopy<<<blocksPerGrid, threadPerBlock>>>(cudaMA, cudaMB, row, column);
  hipDeviceSynchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize( stop );

  /* Calculating run time. */
  hipEventElapsedTime( &time, start, stop );
  hipEventDestroy( start );
  hipEventDestroy( stop );
  printf("%f\n", time);


  if (hipMemcpy(MatrixB, cudaMB, size, hipMemcpyDeviceToHost) != hipSuccess)
      printf("Erro na cópia do Device para o Host!\n");

  
  // for (i = 0; i < row; i++) {
  //   for (j = 0; j < column; j++) {
  //     printf("%.lf\n", MatrixB[i*column+j]);
  //   }
  // }
  
  hipFree(&cudaMA);
  hipFree(&cudaMB);

  return 0;
}
