#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

__global__ void MatrixCopy (double* MatrixA, double* MatrixB, int row, int column) {
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.x*blockDim.x+threadIdx.y;

  MatrixB[i+j*column] = MatrixA[i+j*column];
}

int main () {
  int row = 32,
      column = 32,
      i = 0,
      j = 0;
  dim3 threadPerBlock(16, 16),
       blocksPerGrid(row/threadPerBlock.x, column/threadPerBlock.y);
  size_t size = row*column*sizeof(double);
  double *MatrixA, *MatrixB, *cudaMA, *cudaMB;
  
  /* Matrix allocation. */
  MatrixA = (double*)malloc(size);
  MatrixB = (double*)malloc(size);
  
  for (i = 0; i < column; i++)
    for (j = 0; j < row; j++) {
      MatrixA[i+j*row] = i+j*row;
      MatrixB[i+j*row] = 0.0;
    }
 
  /* Cuda memory allocation. */
  if (hipMalloc(&cudaMA, size) != hipSuccess)
      printf("Erro na alçocação de recursos!\n");
  if (hipMalloc(&cudaMB, size) != hipSuccess)
      printf("Erro na alçocação de recursos!\n");

  /* Cuda memory copy. */
  if (hipMemcpy(cudaMA, MatrixA, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");
  if (hipMemcpy(cudaMB, MatrixB, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");

  /* Cuda kernel call. */
  MatrixCopy<<<blocksPerGrid, threadPerBlock>>>(cudaMA, cudaMB, row, column);
  hipDeviceSynchronize();

  if (hipMemcpy(MatrixB, cudaMB, size, hipMemcpyDeviceToHost) != hipSuccess)
      printf("Erro na cópia do Device para o Host!\n");

  for (i = 0; i < column; i++) {
    for (j = 0; j < row; j++) {
      printf("%5.lf ", MatrixB[i+j*row]);
    }
    printf("\n");
  }
  
  hipFree(&cudaMA);
  hipFree(&cudaMB);

  return 0;
}
