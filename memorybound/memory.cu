#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

__global__ void MatrixCopy (double* MatrixA, double* MatrixB, int row, int column) {
  int i = threadIdx.x;
  int j = threadIdx.y;

  MatrixB[i+j*row] = MatrixA[i+j*row];
}

int main () {
  int row = 10,
      column = 10,
      i = 0,
      j = 0;
  int blocksPerGrid = 1;
  dim3 threadPerBlock(row, column);
  size_t size = row*column*sizeof(double);
  double *MatrixA, *MatrixB, *cudaMA, *cudaMB;

  MatrixA = (double*)malloc(size);
  MatrixB = (double*)malloc(size);
  
  for (i = 0; i < column; i++)
    for (j = 0; j < row; j++) {
      MatrixA[i+j*row] = i+j*row;
      MatrixB[i+j*row] = 0.0;
    }
  
  /* Cuda memory allocation. */
  hipMalloc(&cudaMA, size);
  hipMalloc(&cudaMB, size);

  /* Cuda memory copy. */
  hipMemcpy(cudaMA, MatrixA, size, hipMemcpyHostToDevice);
  hipMemcpy(cudaMB, MatrixB, size, hipMemcpyHostToDevice);

  /* Cuda kernel call. */
  MatrixCopy<<<blocksPerGrid, threadPerBlock>>>(cudaMA, cudaMB, row, column);

  hipMemcpy(MatrixB, cudaMB, size, hipMemcpyDeviceToHost);

  for (i = 0; i < column; i++) {
    for (j = 0; j < row; j++) {
      printf("%lf\t", MatrixB[i+j*row]);
    }
    printf("\n");
  }

  return 0;
}
