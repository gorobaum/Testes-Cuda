#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

__global__ void MatrixCopy (double* MatrixA, double* MatrixB, int row, int column) {
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  MatrixB[i*column+j] = MatrixA[i*column+j];
}

int main () {
  int row = 32,
      column = 32,
      i = 0,
      j = 0;
  dim3 threadPerBlock(16, 16),
       blocksPerGrid(row/threadPerBlock.x+1, column/threadPerBlock.y+1);
  size_t size = row*column*sizeof(double);
  double *MatrixA, *MatrixB, *cudaMA, *cudaMB;
  float time;
  hipEvent_t start, stop;
  
  /* Matrix allocation. */
  MatrixA = (double*)malloc(size);
  MatrixB = (double*)malloc(size);
  
  for (i = 0; i < row; i++)
    for (j = 0; j < column; j++) {
      MatrixA[i*column+j] = i*column+j;
      MatrixB[i*column+j] = 0.0;
    }
 
  /* Cuda memory allocation. */
  if (hipMalloc(&cudaMA, size) != hipSuccess)
      printf("Erro na alçocação de recursos!\n");
  if (hipMalloc(&cudaMB, size) != hipSuccess)
      printf("Erro na alçocação de recursos!\n");

  /* Cuda memory copy. */
  if (hipMemcpy(cudaMA, MatrixA, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");
  if (hipMemcpy(cudaMB, MatrixB, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");
  
  /* Cuda time counter init. */
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Cuda kernel call. */
  hipEventRecord(start, 0);
  MatrixCopy<<<blocksPerGrid, threadPerBlock>>>(cudaMA, cudaMB, row, column);
  hipDeviceSynchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize( stop );

  /* Calculating run time. */
  hipEventElapsedTime( &time, start, stop );
  hipEventDestroy( start );
  hipEventDestroy( stop );
  printf("Total run time on GPU = %fms\n", time);


  if (hipMemcpy(MatrixB, cudaMB, size, hipMemcpyDeviceToHost) != hipSuccess)
      printf("Erro na cópia do Device para o Host!\n");

  /*
  for (i = 0; i < row; i++) {
    for (j = 0; j < column; j++) {
      printf("%5.lf ", MatrixB[i*column+j]);
    }
    printf("\n");
  }*/
  
  hipFree(&cudaMA);
  hipFree(&cudaMB);

  return 0;
}
