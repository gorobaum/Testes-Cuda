#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

__global__ void MatrixCopy (float* MatrixA, float* MatrixB, int rowSize, int columnSize) {
  int row = blockIdx.x*blockDim.x+threadIdx.x;
  int column = blockIdx.y*blockDim.y+threadIdx.y;
  MatrixB[row+column*columnSize] = MatrixA[row+column*columnSize];
}

int main () {
  int row = 4096,
      column = 4096,
      i = 0,
      j = 0;
  dim3 threadPerBlock(16, 16),
       blocksPerGrid(row/threadPerBlock.x, column/threadPerBlock.y);
  size_t size = row*column*sizeof(float);
  float *MatrixA, *MatrixB, *cudaMA, *cudaMB;
  float time;
  hipEvent_t start, stop;
  
  /* Matrix allocation. */
  MatrixA = (float*)malloc(size);
  MatrixB = (float*)malloc(size);
  
  for (i = 0; i < row; i++)
    for (j = 0; j < column; j++) {
      MatrixA[i*column+j] = i+j;
    }
 
  /* Cuda time counter init. */
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Cuda memory allocation. */
  if (hipMalloc(&cudaMA, size) != hipSuccess)
      printf("Erro na alocação de recursos!\n");
  if (hipMalloc(&cudaMB, size) != hipSuccess)
      printf("Erro na alocação de recursos!\n");

  /* Cuda memory copy. */
  if (hipMemcpy(cudaMA, MatrixA, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");
  if (hipMemcpy(cudaMB, MatrixB, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("Erro na cópia de recursos!\n");

  /* Cuda kernel call. */
  hipEventRecord(start, 0);
  MatrixCopy<<<blocksPerGrid, threadPerBlock>>>(cudaMA, cudaMB, row, column);
  hipDeviceSynchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize( stop );

  /* Calculating run time. */
  hipEventElapsedTime( &time, start, stop );
  hipEventDestroy( start );
  hipEventDestroy( stop );
  printf("%f\n", time);


  if (hipMemcpy(MatrixB, cudaMB, size, hipMemcpyDeviceToHost) != hipSuccess)
      printf("Erro na cópia do Device para o Host!\n");

  
  /*for (i = 0; i < row; i++) {
    for (j = 0; j < column; j++) {
      printf("%.lf\n", MatrixB[i*column+j]);
    }
  }*/
  
  hipFree(&cudaMA);
  hipFree(&cudaMB);
  free(MatrixA);
  free(MatrixB);

  return 0;
}
