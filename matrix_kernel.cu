#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "matrix_kernel.h"

__global__ void multi_kernel(float *ma, float *mb, float *mc) {
  int row = threadIdx.x;
  int column = threadIdx.y;
  int i = 0;
  
  row *= MS;
  mc[row+column] = 0;
  
  for (i = 0; i < MS; i++) {
    mc[row+column] += ma[row+i]*mb[column+i*MS];
  }
}

void matrixMulti_caller(float **Ma, float **Mb, float *Mc) {
  float *cudamtxa, *cudamtxb, *cudamtxc;
  dim3 threadPerBlock(MS,MS);

  hipMalloc(&cudamtxa, MS*MS*sizeof(float));
  hipMalloc(&cudamtxb, MS*MS*sizeof(float));
  hipMalloc(&cudamtxc, MS*MS*sizeof(float));

  hipMemcpy(cudamtxa, Ma, MS*MS*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(cudamtxb, Mb, MS*MS*sizeof(float), hipMemcpyHostToDevice);

  multi_kernel<<<1, threadPerBlock>>>(cudamtxa, cudamtxb, cudamtxc);

  printf("Matrixc[0][0] = %f\n", Ma[1][1]);

  hipMemcpy(Mc, cudamtxc, MS*MS*sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  printf("Matrixc[0][0] = %f\n", Mc[11]);
}
