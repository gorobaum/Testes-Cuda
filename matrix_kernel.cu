#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "matrix_kernel.h"

__global__ void multi_kernel(float *ma, float *mb, float *mc) {
  int row = threadIdx.x;
  int column = threadIdx.y;
  row *= MS;
  mc[row+column] = 0;
  int i = 0;
  for (i = 0; i < MS; i++) {
    mc[row+column] += ma[row+i]*mb[column+i*MS];
  }
}

void matrixMulti_caller(float **Ma, float **Mb, float ***Mc) {
  float *cudamtxa, *cudamtxb, *cudamtxc;
  size_t  size = MS*MS*sizeof(float);
  dim3 threadPerBlock(MS,MS);


  hipMalloc(&cudamtxa, size);
  hipMalloc(&cudamtxb, size);
  hipMalloc(&cudamtxc, size);

  hipMemcpy(cudamtxa, Ma, size, hipMemcpyHostToDevice);
  hipMemcpy(cudamtxb, Mb, size, hipMemcpyHostToDevice);

  multi_kernel<<<1, threadPerBlock>>>(cudamtxa, cudamtxb, cudamtxc);

  hipMemcpy((*Mc), cudamtxc, size, hipMemcpyDeviceToHost);

}
